#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>

#define BLOCK_SIZE 4


void read_from_file(int* matrix, const char* path, int size_y, int size_x)
// have you ever heard who is error handling for?
{
    FILE* f;

//     f = fopen(path, "rb");
//     fread(matrix, SIZE_Y*SIZE_X, sizeof(int), f);

// non binary files for now
    f = fopen(path, "r");
    for (int i = 0; i < size_x*size_y; i++)
        fscanf(f, "%d ", (matrix + i));

    fclose(f);
    return;
}

__global__ void conv(float* kernel, int ker_size_y, int ker_size_x, int* matrix, int size_x, int res_size_y, int res_size_x, int* convolved_matrix)
{
    float sum = 0;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if ((y < res_size_y) and (x < res_size_x)) // the image size may not be a multiple of BLOCK_SIZE, therefore an extra check is performed
    {
        printf("You are here (%d, %d)\n", x, y);
        for (int i = 0; i < ker_size_y; i++)
            for (int j = 0; j < ker_size_x; j++)
                sum += *(kernel + i * ker_size_x + j) * *(matrix + (y + i) * size_x + x + j);
        printf("Are you here yet? (%d, %d)\n", x, y);
        convolved_matrix[y * res_size_x + x] = static_cast<int>(sum); // is it good to do this in terms of efficiency?
                                                                      // should i use round() from <cmath> instead?
    }
    printf("You're not here (%d, %d)\n", x, y);
    return;
}

void save_matrix(int* matrix, const char* path, int res_size_y, int res_size_x)
{
    FILE* f;
    f = fopen(path, "w");
    for (int i = 0; i < res_size_y; i++)
    {
        for (int j = 0; j < res_size_x; j++)
        {
            fprintf(f, "%d ", *(matrix + i * res_size_x + j));
        }
        fprintf(f, "\n");
    }
    fclose(f);
    return;
}

int main()
{
// events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
// kernel parameters and the kernel itself
    int ker_size_y = 1;
    int ker_size_x = 1;
    float kernel[ker_size_y * ker_size_x] = {1};
// host: the matrix and convolved matrix
    int size_x = 10;
    int size_y = 10;
    int* matrix = (int *)malloc(size_y * size_x * sizeof(int));

    int res_size_y = size_y - ker_size_y + 1;
    int res_size_x = size_x - ker_size_x + 1;
    printf("res_x = %d; res_y = %d\n", res_size_x, res_size_y);
    int* convolved_matrix = (int *)malloc(res_size_y * res_size_x * sizeof(int));

    read_from_file(matrix, "matrix.txt", size_y, size_x);
// device: same
    int *d_matrix = NULL;
    hipMalloc((void **)&d_matrix, size_y * size_x * sizeof(int));

    int *d_convolved_matrix = NULL;
    hipMalloc((void **)&d_convolved_matrix, res_size_y * res_size_x * sizeof(int));
// copy matrix
    hipMemcpy(d_matrix, matrix, size_y * size_x * sizeof(int), hipMemcpyHostToDevice);
// calculating grid and block parameters
    dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    int bpg_x = res_size_x / BLOCK_SIZE; // bpg goes for "blocks per grid"
    if (res_size_x % BLOCK_SIZE != 0)
        bpg_x++;
    int bpg_y = res_size_y / BLOCK_SIZE;
    if (res_size_y % BLOCK_SIZE != 0)
        bpg_y++;
    dim3 blocksPerGrid = dim3(bpg_x, bpg_y);
//     printf("shape of res = (%d, %d)\nblocksPerGrid = dim3(%d, %d)\n", res_size_x, res_size_y, bpg_x, bpg_y);

    hipEventRecord(start, 0);
// magic
    conv<<<blocksPerGrid, threadsPerBlock>>>(kernel, ker_size_y, ker_size_x, d_matrix, size_x, res_size_y, res_size_x, d_convolved_matrix);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float KernelTime;
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("KernelTime: %.2f milliseconds\n", KernelTime);
// copy convolved matrix
    hipMemcpy(convolved_matrix, d_convolved_matrix, res_size_y * res_size_x * sizeof(int), hipMemcpyDeviceToHost);

    save_matrix(convolved_matrix, "conv_matrix.txt", res_size_y, res_size_x);

    hipFree(d_matrix);
    hipFree(d_convolved_matrix);
    free(matrix);
    free(convolved_matrix);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
